#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#include <sys/time.h>

// 두 시각 사이의 차를 구하기 위한 함수
void getGapTime(struct timeval* start_time, struct timeval* end_time, struct timeval* gap_time)
{
    gap_time->tv_sec = end_time->tv_sec - start_time->tv_sec;
    gap_time->tv_usec = end_time->tv_usec - start_time->tv_usec;
    if(gap_time->tv_usec < 0){
        gap_time->tv_usec = gap_time->tv_usec + 1000000;
        gap_time->tv_sec -= 1;
    }
}

// 시간 보정을 위한 함수
float timevalToFloat(struct timeval* time){
    double val;
    val = time->tv_sec;
    val += (time->tv_usec * 0.000001);
    return val;
}

// Jpg 파일을 그레이 스케일하는 커널 함수
__global__ void JpgToGray(unsigned char *img, int width, int height)
{
    unsigned char* pixelOffset;
    unsigned bytePerPixel = 3;

    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    // (i, j)번째 픽셀에 접근. 해당 픽셀은 R, G, B로 구성.
    if (tid < width * height)
    {
        pixelOffset = img + tid * bytePerPixel;
        unsigned char r = pixelOffset[0];
        unsigned char g = pixelOffset[1]; 
        unsigned char b = pixelOffset[2];
        unsigned char gray = r * 0.2126 + g * 0.7152 + b * 0.0722; // 그레이 스케일을 위한 공식 적용
        pixelOffset[0] = gray;
        pixelOffset[1] = gray;
        pixelOffset[2] = gray;
    }
}

int main(int argc, char *argv[])
{
    // 메모리 복사 및 GPU 연산 시작과 끝의 시간 저장을 위한 구조체 선언
    struct timeval htod_start, htod_end;
    struct timeval gpu_start, gpu_end;
    struct timeval dtoh_start, dtoh_end;

    // stb 라이브러리를 사용해 Jpg를 배열로 로드. Jpg 파일 이름은 argv[1]으로 받음.
    int width, height, channels;
    unsigned bytePerPixel = 3;
    unsigned char *img = stbi_load(argv[1], &width, &height, &channels, 0);
    unsigned char *d_img;

    int size = width*height*bytePerPixel*sizeof(char);
    int totalThread = width*height;
    int blockCount, threadCount;
    threadCount = 1024;
    blockCount = totalThread / threadCount;

    // 디바이스에 배열을 위한 메모리 할당
    hipMalloc((void **)&d_img, size);

    // 호스트에서 디바이스로 배열 복사
    gettimeofday(&htod_start, NULL);
    hipMemcpy(d_img, img, size, hipMemcpyHostToDevice);
    gettimeofday(&htod_end, NULL);

    // 그레이스케일 함수 호출
    gettimeofday(&gpu_start, NULL);
    JpgToGray<<<blockCount+1, threadCount>>>(d_img, width, height);
    // JpgToGray<<<65535,512>>(argv[1], argv[2]);
    hipDeviceSynchronize();
    gettimeofday(&gpu_end, NULL);

    // 디바이스에서 호스트로 배열 복사
    gettimeofday(&dtoh_start, NULL);
    hipMemcpy(img, d_img, size, hipMemcpyDeviceToHost);
    gettimeofday(&dtoh_end, NULL);

    // stb 라이브러리를 사용해 배열을 Jpg로 저장. Jpg 파일 이름은 argv[2]으로 받음.
    stbi_write_jpg(argv[2], width, height, channels, img, 95);

    // 두 시각 사이의 차이 계산 및 출력
    struct timeval htod_gap, gpu_gap, dtoh_gap;
    getGapTime(&htod_start, &htod_end, &htod_gap);
    getGapTime(&gpu_start, &gpu_end, &gpu_gap);
    getGapTime(&dtoh_start, &dtoh_end, &dtoh_gap);
    
    float f_htod_gap = timevalToFloat(&htod_gap);
    float f_gpu_gap = timevalToFloat(&gpu_gap);
    float f_dtoh_gap = timevalToFloat(&dtoh_gap);
    float total_gap = f_htod_gap + f_gpu_gap + f_dtoh_gap;

    printf("GPU 버전 연산 시간 : %.6f + %.6f + %.6f = %.6f\n", f_htod_gap, f_gpu_gap, f_dtoh_gap, total_gap);

    stbi_image_free(img);
    hipFree(d_img);

    return 0;
}